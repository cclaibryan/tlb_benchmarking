#include "hip/hip_runtime.h"
#include <iostream>
#include <set>
#include <algorithm>
#include <assert.h>
#include "hip/hip_runtime.h"
using namespace std;

#define ITERATION_FINEGRAINED   (1)
#define KB                      (1024/sizeof(int))
#define MB                      (KB*1024)
#define MAX_NUM_THREADS         (1024)      // a block has maximal thread size

//kernel function
__global__ void strided_access_onepass(unsigned *arr, int length, int stride, bool record, unsigned *duration, double *help);   //used to benchmark the TLB structure
__global__ void strided_access_finegrained(unsigned *arr, int length, bool record, unsigned *duration, unsigned *index);        //obsolete: use to attain average cycle and pages visited

void TLB_latency(int N, int stride);
void TLB_benchmarking(int beginSize, int endSize, int stride);

void generate_strided(unsigned *arr, int length, int stride);
void generate_strided_onepass(unsigned *arr, int length, int stride);

//global variables
int numThreadsGlobal;
int numBlocksGlobal;
int dataSizeGlobal;   //in MB
int pageSizeGlobal;   //in KB

/*
 * TLB benchmarking: ./tlb_GPU pageSize_KB dataSize_begin_MB dataSize_end_MB
 *
 * blockSize=1 and gridSize=1 for TLB benchmarking;
 */
int main(int argc, char* argv[]){

    if (argc < 4)  {
        cerr<<"Shall provide the blockSize, gridSize used and page size."<<endl;
        cerr<<"Eg.: ./tlb_GPU bSize gSize dataSize_MB pageSize_KB"<<endl;
        exit(0);
    }

    numThreadsGlobal = 1;
    numBlocksGlobal = 1;
    pageSizeGlobal = atoi(argv[1]) * KB;
    int dataSize_begin = atoi(argv[2]) * MB;
    int dataSize_end = atoi(argv[3]) * MB;
    hipSetDevice(0);

    TLB_benchmarking(dataSize_begin, dataSize_end,pageSizeGlobal);

    hipDeviceReset();
    return 0;
}

void TLB_benchmarking(int beginSize, int endSize, int stride) {

    for (int ds = beginSize; ds <= endSize; ds += stride) {
        cout << "Struc: Data size: " << (float)ds / MB << "MB\t" << "Stride: " << stride / MB << "MB\t";

        hipDeviceReset();
        hipError_t error_id;
        unsigned *h_a, *d_a;
        unsigned *h_timeinfo, *d_timeinfo;
        double *help;

        h_a = (unsigned*)malloc(sizeof(unsigned)*ds);
        error_id = hipMalloc ((void **) &d_a, sizeof(unsigned)*ds);
        if (error_id != hipSuccess)    cerr<<"Error 1.0 is "<<hipGetErrorString(error_id)<<endl;

        /* initialize array elements on CPU with pointers into d_a. */
        generate_strided_onepass(h_a,ds,stride);

        /* copy array elements from CPU to GPU */
        error_id = hipMemcpy(d_a, h_a, sizeof(unsigned)*ds, hipMemcpyHostToDevice);
        if (error_id != hipSuccess)    cerr<<"Error 1.1 is "<<hipGetErrorString(error_id)<<endl;

        h_timeinfo = (unsigned *) malloc(sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal);
        error_id = hipMalloc((void **) &d_timeinfo, sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal);
        if (error_id != hipSuccess)    cerr << "Error 1.2 is " << hipGetErrorString(error_id) << endl;

        error_id = hipMalloc((void **) &help, sizeof(double) * numThreadsGlobal * numBlocksGlobal);
        if (error_id != hipSuccess)    cerr << "Error 1.3 is " << hipGetErrorString(error_id) << endl;

        hipDeviceSynchronize();
        /* launch kernel*/
        dim3 Db = dim3(numThreadsGlobal);
        dim3 Dg = dim3(numBlocksGlobal);

        strided_access_onepass<<< Dg, Db >>> (d_a, ds, stride, false, NULL, NULL);        //warp up
        strided_access_onepass<<< Dg, Db >>> (d_a, ds, stride, true, d_timeinfo, help);   //formal

        hipDeviceSynchronize();

        error_id = hipGetLastError();
        if (error_id != hipSuccess) {
            cerr << "Error kernel is " << hipGetErrorString(error_id) << endl;
        }

        /* copy results from GPU to CPU */
        hipDeviceSynchronize();

        error_id = hipMemcpy((void *) h_timeinfo, (void *) d_timeinfo, sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal, hipMemcpyDeviceToHost);
        if (error_id != hipSuccess)    cerr << "Error 2.2 is " << hipGetErrorString(error_id) << endl;

        double total = 0;       //here we use double, otherwise it will overflow
        for (int i = 0; i < numThreadsGlobal * numBlocksGlobal; i++) {
            total += h_timeinfo[i];
        }
        total /= (numThreadsGlobal * numBlocksGlobal);
        cout << "cycle: " << total << endl;

        hipDeviceSynchronize();

        /* free memory on GPU */
        hipFree(help);
        hipFree(d_a);
        hipFree(d_timeinfo);

        /*free memory on CPU */
        free(h_a);
        free(h_timeinfo);

        hipDeviceReset();
    }
}

//used for TLB benchmarking
__global__ void strided_access_onepass(unsigned *arr, int length, int stride, bool record, unsigned *duration, double *help) {

    unsigned long start, end;
    unsigned gid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned curIdx = 0;

    double anc = 0;
    double total = 0;
    int myIteration = 0;

    //traverse the data array once
    while (curIdx < length) {
        start = clock64();
            curIdx = arr[curIdx];
            anc += curIdx;                  //to ensure the curIdx has been read, this instruction is 16-cycle long on K40m
        end = clock64();
        total += (end-start-16);
        myIteration++;
    }

    if (record)     {
        duration[gid] = (total/myIteration);
        help[gid] = anc;
    }
}

void generate_strided_onepass(unsigned *arr, int length, int stride) {
    for (int i = 0 ; i < length; i++) {
        arr[i] = i+stride;
    }
}

//void measure_global() {
//
//    int stride = pageSizeGlobal*KB; //2MB stride
//    set<int> missPages; //recording the overall missing pages in each case
//
//    //begin and end size in MBs
//    /* To test the TLB structures the beginSize and endSize is different;
//     * To test the latency of multi-thread, beginSize and endSize should set as the data size tested */
//    int beginSize = dataSizeGlobal * MB;
//    int endSize = dataSizeGlobal * MB;
//
//    //1. The L1 TLB has 16 entries. Test with N_min=28 *1024*256, N_max>32*1024*256
//    //2. The L2 TLB has 65 entries. Test with N_min=128*1024*256, N_max=160*1024*256
//    for (int dataSize = beginSize; dataSize <= endSize; dataSize += (128*KB)) {
////        cout<<"Data size: "<<(float)dataSize/MB<<"MB\t"<<"Stride: "<< stride/MB <<"MB"<<endl;
//        cout<<"Data size: "<<(float)dataSize/MB<<"MB\tbsize: "<<numThreadsGlobal<<"\tgsize: "<<numBlocksGlobal<<'\t';
//        parametric_measure_global(dataSize, false, stride, missPages);  //not finegrained
//    }
//}
//void TLB_finegrained(int N, bool finegrained, int stride, set<int> & lastMissPages) {
//    hipDeviceReset();
//    hipError_t error_id;
//    int i;
//    unsigned *h_a, *d_a;
//    h_a = (unsigned*)malloc(sizeof(unsigned)*N);
//    error_id = hipMalloc ((void **) &d_a, sizeof(unsigned)*N);
//
//    if (error_id != hipSuccess)
//        cerr<<"Error 1.0 is "<<hipGetErrorString(error_id)<<endl;
//
//    /* initialize array elements on CPU with pointers into d_a. */
//    generate_strided(h_a,N,stride);
//    //generate_strided_onepass(h_a,N,(mul)*stride);
//
//    /* copy array elements from CPU to GPU */
//    error_id = hipMemcpy(d_a, h_a, sizeof(unsigned)*N, hipMemcpyHostToDevice);
//    if (error_id != hipSuccess) {
//        cerr<<"Error 1.1 is "<<hipGetErrorString(error_id)<<endl;
//    }
//
//    unsigned  *h_index, *h_timeinfo, *d_timeinfo, *d_index;
//    double *help;
//
//    if (finegrained) {
//        h_index = (unsigned *) malloc(sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal * ITERATION);
//        h_timeinfo = (unsigned *) malloc(sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal * ITERATION);
//
//        //recording time and visited locations
//        error_id = hipMalloc((void **) &d_timeinfo, sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal * ITERATION);
//        if (error_id != hipSuccess) {
//            cerr << "Error 1.2 is " << hipGetErrorString(error_id) << endl;
//        }
//
//        error_id = hipMalloc((void **) &d_index, sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal *ITERATION);
//        if (error_id != hipSuccess) {
//            cerr << "Error 1.3 is " << hipGetErrorString(error_id) << endl;
//        }
//    }
//    else {
//        h_timeinfo = (unsigned *) malloc(sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal);
//        error_id = hipMalloc((void **) &d_timeinfo, sizeof(unsigned) * numThreadsGlobal * numBlocksGlobal);
//        if (error_id != hipSuccess) {
//            cerr << "Error 1.4 is " << hipGetErrorString(error_id) << endl;
//        }
//        error_id = hipMalloc((void **) &help, sizeof(double) * numThreadsGlobal * numBlocksGlobal);
//        if (error_id != hipSuccess) {
//            cerr << "Error 1.5 is " << hipGetErrorString(error_id) << endl;
//        }
//    }
//
//    hipDeviceSynchronize ();
//    /* launch kernel*/
//    dim3 Db = dim3(numThreadsGlobal);
//    dim3 Dg = dim3(numBlocksGlobal);
//    if (finegrained) {
//        strided_access_finegrained<<<Dg, Db>>>(d_a, N, false, NULL, NULL);
//        strided_access_finegrained<<<Dg, Db>>>(d_a, N, false, d_timeinfo, d_index);
//    }
//    else {
//        strided_access<<<Dg, Db>>>(d_a, N, stride, false, NULL, NULL);        //warp up
//        strided_access<<<Dg, Db>>>(d_a, N, stride, true, d_timeinfo, help);   //formal
//    }
//
//    hipDeviceSynchronize();
//
//    error_id = hipGetLastError();
//    if (error_id != hipSuccess) {
//        cerr<<"Error kernel is "<<hipGetErrorString(error_id)<<endl;
//    }
//
//    /* copy results from GPU to CPU */
//    hipDeviceSynchronize ();
//
//    if (finegrained) {
//        error_id = hipMemcpy((void *)h_timeinfo, (void *)d_timeinfo, sizeof(unsigned)*ITERATION*numThreadsGlobal * numBlocksGlobal, hipMemcpyDeviceToHost);
//        if (error_id != hipSuccess) {
//            cerr<<"Error 2.0 is "<<hipGetErrorString(error_id)<<endl;
//        }
//        error_id = hipMemcpy((void *)h_index, (void *)d_index, sizeof(unsigned)*ITERATION*numThreadsGlobal * numBlocksGlobal, hipMemcpyDeviceToHost);
//        if (error_id != hipSuccess) {
//            cerr<<"Error 2.1 is "<<hipGetErrorString(error_id)<<endl;
//        }
//
//        //statistics
//        int count_less_300 = 0, count_300_400 = 0, count_400_500 = 0, count_500_600 = 0, count_larger_600 = 0;
//        double total = 0;
//
//        int loop = 0;           //how many times the array is looped
//
//        set<int> curMissPages;
//        for(i=0 ;i<ITERATION;i++) {
//            int curPage = h_index[i]/stride;
//            if ( (h_timeinfo[i] > 400) && (h_timeinfo[i] < 510)) {
//                curMissPages.insert(curPage);
//            }
//            cout<<curPage<<'\t'<<h_index[i]<<'\t'<<h_timeinfo[i]<<endl;
//
//            if (h_index[i]<stride)  loop ++;
//            if (h_timeinfo[i] < 300)            count_less_300++;
//            else if (h_timeinfo[i] < 400) count_300_400 ++;
//            else if (h_timeinfo[i] < 500) count_400_500 ++;
//            else if (h_timeinfo[i] < 600) count_500_600++;
//            else                                                    count_larger_600++;
//            total += h_timeinfo[i];
//        }
//        set<int> diffSet;
//        set_difference(curMissPages.begin(), curMissPages.end(),lastMissPages.begin(), lastMissPages.end(), inserter(diffSet,diffSet.end()));
//
//        //to check that pages missed in last dataset will be hit in this dataset
//        set<int> checkSet;
//        set_difference(lastMissPages.begin(), lastMissPages.end(), curMissPages.begin(), curMissPages.end(), inserter(checkSet,checkSet.end()));
//        assert(checkSet.size() == 0);
//
//        int totalPages = N /512 / 1024;
//        cout<<"Pages: "<<totalPages<<", misses: "<<count_400_500<<", loops: "<<loop<<", new miss pages: ";
//        for (set<int>::iterator it = diffSet.begin(); it != diffSet.end(); ++it) {
//            cout<<*it<<' ';
//        }
//        cout<<endl;
//
//        // lastMissPages = curMissPages;
//
//        total = total / ITERATION;
//        cout<<"Average: "<<total<<endl;
//        // cout<<"Statistics:"<<endl;
//        // cout<<"Data size: "<<N / 1024 / 256<<" MB."<<endl;
//
//        // cout<<"less than 300: "<<count_less_300<<endl;
//        // cout<<"300 - 400: "<<count_300_400<<endl;
//        // cout<<"400 - 500: "<<count_400_500<<endl;
//        // cout<<"500 - 600: "<<count_500_600<<endl;
//        // cout<<"larger than 600: "<<count_larger_600<<endl;
//        // cout<<"Average cycles: "<<total<<" in "<<ITERATION<<" iterations."<<endl;
//    }
//    else {
//        error_id = hipMemcpy((void *)h_timeinfo, (void *)d_timeinfo, sizeof(unsigned)*numThreadsGlobal * numBlocksGlobal, hipMemcpyDeviceToHost);
//        if (error_id != hipSuccess) {
//            cerr<<"Error 2.2 is "<<hipGetErrorString(error_id)<<endl;
//        }
//
//        double total = 0;       //here we use double, otherwise it will overflow
//        for(int i = 0; i < numThreadsGlobal*numBlocksGlobal; i++) {
//            total += h_timeinfo[i];
//        }
//        total /= (numThreadsGlobal*numBlocksGlobal);
//        cout<<"cycle: "<<total<<endl;
//    }
//    hipDeviceSynchronize();
//
//    /* free memory on GPU */
//    if (finegrained) {
//        hipFree(d_index);
//        free(h_index);
//    }
//    else {
//        hipFree(help);
//    }
//
//    hipFree(d_a);
//    hipFree(d_timeinfo);
//
//    /*free memory on CPU */
//    free(h_a);
//    free(h_timeinfo);
//
//    hipDeviceReset();
//}
//obsolete: to record the page number and study the cache replacement policy
//__global__ void strided_access_finegrained(unsigned *arr, int length, bool record, unsigned *duration, unsigned *index) {
//
//    unsigned timestamp;
//    unsigned gid = blockDim.x * blockIdx.x + threadIdx.x;
//    unsigned gsize = blockDim.x * gridDim.x;
//    unsigned curIdx = (blockDim.x * threadIdx.x + blockIdx.x) % length;
//
//    __shared__ unsigned int s_tvalue[ITERATION_FINEGRAINED*MAX_NUM_THREADS];
//    __shared__ unsigned int s_index[ITERATION_FINEGRAINED*MAX_NUM_THREADS];
//
//    unsigned it = gid;
//    while (it < ITERATION_FINEGRAINED * MAX_NUM_THREADS) {
//        s_index[it] = 0;
//        s_tvalue[it] = 0;
//        it += gsize;
//    }
//    __syncthreads();
//
//    it = gid;
//    for (int k = 0; k < ITERATION_FINEGRAINED; k++) {
//        timestamp = clock();
//        curIdx = arr[curIdx];
//        s_index[it]= curIdx;
//        timestamp = clock() - timestamp;
//        s_tvalue[it] = timestamp;
//        it += ITERATION_FINEGRAINED;
//    }
//
//    if (record) {
//        it = threadIdx.x;
//        while (it < blockDim.x * ITERATION_FINEGRAINED) {
//            duration[it + blockIdx.x*blockDim.x*ITERATION_FINEGRAINED] = s_tvalue[it];
//            index[it + blockIdx.x*blockDim.x*ITERATION_FINEGRAINED] = s_index[it];
//            it += blockDim.x;
//        }
//    }
//}